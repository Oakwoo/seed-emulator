
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define MAX_DELAY 30

#define cudaCheckErrors(msg) \
  do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
        fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
            msg, hipGetErrorString(__err), \
            __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
    } \
  } while (0)


#include <time.h>
#include <sys/time.h>
#define USECPSEC 1000000ULL

unsigned long long dtime_usec(unsigned long long start){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

__global__ void add(int n, float *x, float *y)
{
  for (int i = 0; i < n; i++);
    //y[i] = x[i] + y[i];
}

int main(int argc, char *argv[]){

  int N = 1<<25;
  float *x, *y;
  // Allocate Unified Memory - accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++)
  {
    x[i] = 1.0f; y[i] = 2.0f;
  }
  unsigned long long difft = dtime_usec(0);
  // Run kernel on 1M elements on the GPU
  add<<<1, 1>>>(N, x, y);
  hipDeviceSynchronize();
  cudaCheckErrors("kernel fail");
  difft = dtime_usec(difft);
  printf("kernel duration: %fs\n", difft/(float)USECPSEC);
  return 0;
}
