#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void compute_kernel(float *data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float val = data[idx];
    for (int i = 0; i < 1e7; ++i) {
        val = sinf(val) + cosf(val);
    }
    data[idx] = val;
}

int main() {
    const int N = 1024 * 256;
    float *d_data;
    hipMalloc(&d_data, N * sizeof(float));

    hipMemset(d_data, 0, N * sizeof(float));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    compute_kernel<<<N / 256, 256>>>(d_data);
    hipDeviceSynchronize();

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    printf("Run time: %.2f ms\n", ms);

    hipFree(d_data);
    return 0;
}
